#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/DeviceGuard.h>

#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>

#include <vector>
#include <iostream>


//!see https://zhuanlan.zhihu.com/p/80902998


int const threadsPerBlock = sizeof(unsigned long long) * 8;//分块数量

//在gpu上计算IOU
__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;//确定当前block的横纵坐标

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);
  ////求当前block的行长度，如果最后一个block不够除，则取余下的，
  //比如ceil(105/25) = 5，105 = 4 * 25 + 5最后一块高为5，此时row_size=5，其余的row_size = 25

  // 共享内存，加速数据读取，
  //同一个block有共享内存，所以先使用共享内存存下当前block全部需要读取的数据
  //(即box的坐标和置信度)然后就不在dev_boxes里面读数据了，而是读share memory里面的数据
  __shared__ float block_boxes[threadsPerBlock * 5];

  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }

  //为了保证线程安全，必须等所有的线程都把数据存到share memory以后，统一开始线程
  __syncthreads();
  // 这个if判断去掉多余的thread，保证余下的块可以被正确执行
  // 每个block里面有row_size个线程
  // 每个线程i，for一个col_size的循环，计算该block里面第i个box和该block中每个列box的IOU
  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;//对角线上的block, //自己跟自己就不要计算IOU了
    }
    for (i = start; i < col_size; i++) {
       //主循环，求该box和所有列box的IOU，如果满足条件，则使用一个mask把该位置1
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;//掩码
      }
    }
    const int col_blocks = THCCeilDiv(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

// boxes is a N x 5 tensor
at::Tensor nms_cuda(const at::Tensor boxes, float nms_overlap_thresh) {

  // Ensure CUDA uses the input tensor device.
  at::DeviceGuard guard(boxes.device());

  using scalar_t = float;
  AT_ASSERTM(boxes.type().is_cuda(), "boxes must be a CUDA tensor");
  auto scores = boxes.select(1, 4);//tensor.select(1, index)等效于tensor[:, index]
  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t);

  int boxes_num = boxes.size(0);

  const int col_blocks = THCCeilDiv(boxes_num, threadsPerBlock);

  scalar_t* boxes_dev = boxes_sorted.data<scalar_t>();

  THCState *state = at::globalContext().lazyInitCUDA(); // TODO replace with getTHCState

  unsigned long long* mask_dev = NULL;
  //THCudaCheck(THCudaMalloc(state, (void**) &mask_dev,
  //                      boxes_num * col_blocks * sizeof(unsigned long long)));

  mask_dev = (unsigned long long*) THCudaMalloc(state, boxes_num * col_blocks * sizeof(unsigned long long));


  // 定义blocks的数量和每个block的线程数
  dim3 blocks(THCCeilDiv(boxes_num, threadsPerBlock),
              THCCeilDiv(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
   // 调用kernel，最后在mask_dev中求出每两个框的IoU是否超过阈值t
  nms_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  THCudaCheck(hipMemcpyAsync(
			  &mask_host[0],
			  mask_dev,
			  sizeof(unsigned long long) * boxes_num * col_blocks,
			  hipMemcpyDeviceToHost,
			  at::cuda::getCurrentCUDAStream()
			  ));

  std::vector<unsigned long long> remv(col_blocks);// 初始是所有框都在S里面，移出标记都置为0
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);// 初始是所有框都在S里面，移出标记都置为0

  at::Tensor keep = at::empty({boxes_num}, boxes.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock; //求这个box是在哪个block里面计算的
    int inblock = i % threadsPerBlock; //求这个box在block的哪个线程计算的
    
    // 对于每个box，如果他在S中，则加入结果集，并移出S
    // 并把和他的IOU大于阈值的所有box全部移出S
    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;//加入结果集操作
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];//移出S操作
      }
    }
  }

  THCudaFree(state, mask_dev);
  // TODO improve this part
  return std::get<0>(order_t.index({
                       keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep).to(
                         order_t.device(), keep.scalar_type())
                     }).sort(0, false));
}
